
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>
#include <cmath>
#include <vector>
#include <bits/stdc++.h>
#include <time.h>


using namespace std;

__global__ void compute(float *d_U_n, float *d_U_n1, int *d_M, int *d_N, float *d_d){
    int t= threadIdx.x+blockDim.x*blockIdx.x;
    d_U_n1[0]=0;
    d_U_n1[*d_N -1]=0;
    if (t < *d_N -2){
        d_U_n1[t+1]= (*d_d)*d_U_n[t]+(1-2*(*d_d))*d_U_n[t+1] +(*d_d)*d_U_n[t+2];
    }

}

int main(){
    int L=1;
    int T=1;
    float dx= 0.1;
    float dt=0.001;
    int alpha=1;
    float d=(alpha*dt)/(dx*dx);

    // space nodes
    int N= L/dx +1;
    cout<<N<<endl;
    // time nodes
    int M= T/dt +1;
    cout<<M<<endl;

    float x[N];
    float t[M];
    for (int i=0; i<N; i++){
        x[i]=0+(i-1)*dx;
    }
    for (int j=0; j<M; j++){
        t[j]= 0+(j-1)*dt;
    }
    float U[M][N];
    for (int i=0; i<M; i++){
        if (i==0){
            for (int j=0; j<N; j++){
                U[i][j]= sin(4*M_PI*x[j]);
            }
        } 

        else{
            for (int j=0; j<N; j++){
                U[i][j]=0;
            }
            
        }
    }
    float *h_U_n;
    h_U_n= (float*)malloc(N*sizeof(float));
    h_U_n[0]=0;
    h_U_n[N-1]=0;
    for (int i=1; i<N-1; i++){
        h_U_n[i-1]=U[0][i];
    }
    float *d_U_n;
    float *d_U_n1;
    int *d_M;
    int *d_N;
    float *d_d;
    
    hipMalloc((void**) &d_M, sizeof(int));
    hipMalloc((void**) &d_N, sizeof(int));
    hipMalloc((void**) &d_d, sizeof(float));

    hipMalloc((void**) &d_U_n, N*sizeof(float));
    hipMalloc((void**) &d_U_n1, N*sizeof(float));
    hipEvent_t start, stop; 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milsec=0.0;
    hipEventRecord(start);
    hipMemcpy(d_M, &M, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_N, &N, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_d, &d, sizeof(float), hipMemcpyHostToDevice);

    for (int i=0; i<M-1; i++){
        hipMemcpy(d_U_n, h_U_n, N*sizeof(float), hipMemcpyHostToDevice);
        compute <<< N/64 +1, 64>>> (d_U_n, d_U_n1, d_M, d_N, d_d);

        hipMemcpy(h_U_n, d_U_n1, N*sizeof(float), hipMemcpyDeviceToHost);
        for (int j=1; j<N-1; j++){
            U[i+1][j]=h_U_n[j];
        }
    }
    hipEventRecord(stop);

    for (int i=0; i<M; i++){
        for (int j=0; j<N; j++){
            cout<<U[i][j]<<" ";
        }
        cout<<endl;
    }
    hipEventElapsedTime(&milsec, start, stop);
    cout<< "Time taken by the GPU is : "<<milsec<<endl;
    
}
